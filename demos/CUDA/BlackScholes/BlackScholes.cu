#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */

/*
 * DISCLAIMER: The following file has been slightly modified to ensure
 * compatibility with Clad and to serve as a Clad demo. Specifically, parts of
 * the original `main` function have been moved to a separate function to use
 * `clad::gradient` on. Furthermore, original print statements have been removed
 * and new helper functions are now included in the file to verify the
 * gradient's results. The original file is available in NVIDIA's cuda-samples
 * repository on GitHub.
 *
 * Relevant documentation regarding the problem at hand can be found in NVIDIA's
 * cuda-samples repository. Using Clad, we compute some of the Greeks
 * (sensitivities) for the Black-Scholes model and verify them against
 * approximations of their theoretical values as denoted in Wikipedia
 * (https://en.wikipedia.org/wiki/Black%E2%80%93Scholes_model).
 *
 * To build and run the demo, use the following command: make run
 */

#include "clad/Differentiator/Differentiator.h"

#include <hip/hip_runtime_api.h> // helper functions CUDA error checking and initialization
#include <helper_functions.h> // helper functions for string parsing

////////////////////////////////////////////////////////////////////////////////
// Process an array of optN options on CPU
////////////////////////////////////////////////////////////////////////////////
extern "C" void BlackScholesCPU(float* h_CallResult, float* h_PutResult,
                                float* h_StockPrice, float* h_OptionStrike,
                                float* h_OptionYears, float Riskfree,
                                float Volatility, int optN);
extern "C" double CND(double d);

////////////////////////////////////////////////////////////////////////////////
// Process an array of OptN options on GPU
////////////////////////////////////////////////////////////////////////////////
#include "BlackScholes_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int OPT_N = 4000000;
const int NUM_ITERATIONS = 512;

const int OPT_SZ = OPT_N * sizeof(float);
const float RISKFREE = 0.02f;
const float VOLATILITY = 0.30f;

#define DIV_UP(a, b) (((a) + (b) - 1) / (b))

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////

void launch(float* h_CallResultCPU, float* h_CallResultGPU,
            float* h_PutResultCPU, float* h_PutResultGPU, float* h_StockPrice,
            float* h_OptionStrike, float* h_OptionYears) {

  //'d_' prefix - GPU (device) memory space
  float
      // Results calculated by GPU
      *d_CallResult = nullptr,
      *d_PutResult = nullptr,
      // GPU instance of input data
          *d_StockPrice = nullptr, *d_OptionStrike = nullptr,
      *d_OptionYears = nullptr;

  hipMalloc((void**)&d_CallResult, OPT_SZ);
  hipMalloc((void**)&d_PutResult, OPT_SZ);
  hipMalloc((void**)&d_StockPrice, OPT_SZ);
  hipMalloc((void**)&d_OptionStrike, OPT_SZ);
  hipMalloc((void**)&d_OptionYears, OPT_SZ);

  // Copy options data to GPU memory for further processing
  hipMemcpy(d_StockPrice, h_StockPrice, OPT_SZ, hipMemcpyHostToDevice);
  hipMemcpy(d_OptionStrike, h_OptionStrike, OPT_SZ, hipMemcpyHostToDevice);
  hipMemcpy(d_OptionYears, h_OptionYears, OPT_SZ, hipMemcpyHostToDevice);

  BlackScholesGPU<<<DIV_UP((OPT_N / 2), 128), 128 /*480, 128*/>>>(
      (float2*)d_CallResult, (float2*)d_PutResult, (float2*)d_StockPrice,
      (float2*)d_OptionStrike, (float2*)d_OptionYears, RISKFREE, VOLATILITY,
      OPT_N);

  // Both call and put is calculated

  // Read back GPU results to compare them to CPU results
  hipMemcpy(h_CallResultGPU, d_CallResult, OPT_SZ, hipMemcpyDeviceToHost);
  hipMemcpy(h_PutResultGPU, d_PutResult, OPT_SZ, hipMemcpyDeviceToHost);

  // Calculate options values on CPU
  BlackScholesCPU(h_CallResultCPU, h_PutResultCPU, h_StockPrice, h_OptionStrike,
                  h_OptionYears, RISKFREE, VOLATILITY, OPT_N);

  hipFree(d_OptionYears);
  hipFree(d_OptionStrike);
  hipFree(d_StockPrice);
  hipFree(d_PutResult);
  hipFree(d_CallResult);
}

double d1(double S, double X, double T) {
  return (log(S / X) + (RISKFREE + 0.5 * VOLATILITY * VOLATILITY) * T) /
         (VOLATILITY * sqrt(T));
}

double N_prime(double d) {
  const double RSQRT2PI =
      0.39894228040143267793994605993438; // 1 / sqrt(2 * PI)
  return RSQRT2PI * exp(-0.5 * d * d);
}

enum Greek { Delta, dX, Theta };

enum OptionType { Call, Put };

template <OptionType opt> const char* getNameofOpt() {
  if constexpr (opt == Call)
    return "Call";
  if constexpr (opt == Put)
    return "Put";
}

template <Greek greek> const char* getNameOfGreek() {
  if constexpr (greek == Delta)
    return "Delta";
  if constexpr (greek == dX)
    return "dStrike";
  if constexpr (greek == Theta)
    return "Theta";
}

template <OptionType opt, Greek greek>
void computeL1norm(float* S, float* X, float* T, float* d) {
  double delta, ref, sum_delta, sum_ref;
  sum_delta = 0;
  sum_ref = 0;
  for (int i = 0; i < OPT_N; i++) {
    if constexpr (opt == Call) {
      if constexpr (greek == Delta) {
        double d1_val = d1(S[i], X[i], T[i]);
        ref = CND(d1_val);
      } else if constexpr (greek == dX) {
        double T_val = T[i];
        double d1_val = d1(S[i], X[i], T_val);
        double d2_val = d1_val - VOLATILITY * sqrt(T_val);
        double expRT = exp(-RISKFREE * T_val);
        ref = -expRT * CND(d2_val);
      } else if constexpr (greek == Theta) {
        double S_val = S[i], X_val = X[i], T_val = T[i];
        double d1_val = d1(S_val, X_val, T_val);
        double d2_val = d1_val - VOLATILITY * sqrt(T_val);
        double expRT = exp(-RISKFREE * T_val);
        ref = (S_val * N_prime(d1_val) * VOLATILITY) / (2 * sqrt(T_val)) +
              RISKFREE * X_val * expRT *
                  CND(d2_val); // theta is with respect to t, so -theta is the
                               // approximation of the derivative with respect
                               // to T
      }
    } else if constexpr (opt == Put) {
      if constexpr (greek == Delta) {
        double d1_val = d1(S[i], X[i], T[i]);
        ref = CND(d1_val) - 1.0;
      } else if constexpr (greek == dX) {
        double T_val = T[i];
        double d1_val = d1(S[i], X[i], T_val);
        double d2_val = d1_val - VOLATILITY * sqrt(T_val);
        double expRT = exp(-RISKFREE * T_val);
        ref = expRT * CND(-d2_val);
      } else if constexpr (greek == Theta) {
        double S_val = S[i], X_val = X[i], T_val = T[i];
        double d1_val = d1(S_val, X_val, T_val);
        double d2_val = d1_val - VOLATILITY * sqrt(T_val);
        double expRT = exp(-RISKFREE * T_val);
        ref = (S_val * N_prime(d1_val) * VOLATILITY) / (2 * sqrt(T_val)) -
              RISKFREE * X_val * expRT * CND(-d2_val);
      }
    }

    delta = fabs(d[i] - ref);
    sum_delta += delta;
    sum_ref += fabs(ref);
  }

  double L1norm = sum_delta / sum_ref;
  printf("L1norm of %s for %s option = %E\n", getNameOfGreek<greek>(),
         getNameofOpt<opt>(), L1norm);
  if (L1norm > 1e-5) {
    printf(
        "Gradient test failed: Difference between %s's computed and "
        "approximated theoretical values for %s option is larger than expected",
        getNameOfGreek<greek>(), getNameofOpt<opt>());
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char** argv) {
  float* h_CallResultCPU = (float*)malloc(OPT_SZ);
  float* h_PutResultCPU = (float*)malloc(OPT_SZ);
  float* h_CallResultGPU = (float*)malloc(OPT_SZ);
  float* h_PutResultGPU = (float*)malloc(OPT_SZ);
  float* h_StockPrice = (float*)malloc(OPT_SZ);
  float* h_OptionStrike = (float*)malloc(OPT_SZ);
  float* h_OptionYears = (float*)malloc(OPT_SZ);

  srand(5347);

  // Generate options set
  for (int i = 0; i < OPT_N; i++) {
    h_CallResultCPU[i] = 0.0f;
    h_PutResultCPU[i] = -1.0f;
    h_StockPrice[i] = RandFloat(5.0f, 30.0f);
    h_OptionStrike[i] = RandFloat(1.0f, 100.0f);
    h_OptionYears[i] = RandFloat(0.25f, 10.0f);
  }

  /*******************************************************************************/

  // Compute gradients
  auto callGrad = clad::gradient(
      launch, "h_CallResultGPU, h_StockPrice, h_OptionStrike, h_OptionYears");
  auto putGrad = clad::gradient(
      launch, "h_PutResultGPU, h_StockPrice, h_OptionStrike, h_OptionYears");

  // Declare and initialize the derivatives
  float* d_CallResultGPU = (float*)malloc(OPT_SZ);
  float* d_PutResultGPU = (float*)malloc(OPT_SZ);
  float* d_StockPrice = (float*)calloc(OPT_N, sizeof(float));
  float* d_OptionStrike = (float*)calloc(OPT_N, sizeof(float));
  float* d_OptionYears = (float*)calloc(OPT_N, sizeof(float));

  for (int i = 0; i < OPT_N; i++) {
    d_CallResultGPU[i] = 1.0f;
    d_PutResultGPU[i] = 1.0f;
  }

  /*******************************************************************************/

  // Compute the values and derivatives of the price of the call options
  callGrad.execute(h_CallResultCPU, h_CallResultGPU, h_PutResultCPU,
                   h_PutResultGPU, h_StockPrice, h_OptionStrike, h_OptionYears,
                   d_CallResultGPU, d_StockPrice, d_OptionStrike,
                   d_OptionYears);

  // Calculate max absolute difference and L1 distance
  // between CPU and GPU results
  double delta, ref, sum_delta, sum_ref, L1norm;
  sum_delta = 0;
  sum_ref = 0;

  for (int i = 0; i < OPT_N; i++) {
    ref = h_CallResultCPU[i];
    delta = fabs(h_CallResultCPU[i] - h_CallResultGPU[i]);
    sum_delta += delta;
    sum_ref += fabs(ref);
  }

  L1norm = sum_delta / sum_ref;
  printf("L1norm = %E\n", L1norm);
  if (L1norm > 1e-6) {
    printf("Original test failed\n");
    return EXIT_FAILURE;
  }

  // Verify delta
  computeL1norm<Call, Delta>(h_StockPrice, h_OptionStrike, h_OptionYears,
                             d_StockPrice);
  // Verify derivatives with respect to the Strike price
  computeL1norm<Call, dX>(h_StockPrice, h_OptionStrike, h_OptionYears,
                          d_OptionStrike);
  // Verify theta
  computeL1norm<Call, Theta>(h_StockPrice, h_OptionStrike, h_OptionYears,
                             d_OptionYears);

  /*******************************************************************************/

  // Re-initialize data for next gradient call
  for (int i = 0; i < OPT_N; i++) {
    h_CallResultCPU[i] = 0.0f;
    h_PutResultCPU[i] = -1.0f;
    d_CallResultGPU[i] = 1.0f;
    d_PutResultGPU[i] = 1.0f;
  }

  for (int i = 0; i < OPT_N; i++) {
    d_StockPrice[i] = 0.f;
    d_OptionStrike[i] = 0.f;
    d_OptionYears[i] = 0.f;
  }

  // Compute the values and derivatives of the price of the Put options
  putGrad.execute(h_CallResultCPU, h_CallResultGPU, h_PutResultCPU,
                  h_PutResultGPU, h_StockPrice, h_OptionStrike, h_OptionYears,
                  d_PutResultGPU, d_StockPrice, d_OptionStrike, d_OptionYears);

  // Verify delta
  computeL1norm<Put, Delta>(h_StockPrice, h_OptionStrike, h_OptionYears,
                            d_StockPrice);
  // Verify derivatives with respect to the Strike price
  computeL1norm<Put, dX>(h_StockPrice, h_OptionStrike, h_OptionYears,
                         d_OptionStrike);
  // Verify theta
  computeL1norm<Put, Theta>(h_StockPrice, h_OptionStrike, h_OptionYears,
                            d_OptionYears);

  /*******************************************************************************/
  free(h_OptionYears);
  free(h_OptionStrike);
  free(h_StockPrice);
  free(h_PutResultGPU);
  free(h_CallResultGPU);
  free(h_PutResultCPU);
  free(h_CallResultCPU);
  free(d_OptionYears);
  free(d_OptionStrike);
  free(d_StockPrice);
  free(d_PutResultGPU);
  free(d_CallResultGPU);

  return EXIT_SUCCESS;
}
