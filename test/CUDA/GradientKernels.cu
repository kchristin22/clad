#include "hip/hip_runtime.h"
// RUN: %cladclang_cuda -I%S/../../include -fsyntax-only \
// RUN:     --cuda-gpu-arch=%cudaarch --cuda-path=%cudapath  -Xclang -verify \
// RUN:     %s 2>&1 | %filecheck %s
//
// RUN: %cladclang_cuda -I%S/../../include --cuda-path=%cudapath \
// RUN:     --cuda-gpu-arch=%cudaarch %cudaldflags -oGradientKernels.out %s
//
// RUN: ./GradientKernels.out | %filecheck_exec %s
//
// REQUIRES: cuda-runtime
//
// expected-no-diagnostics

#include "clad/Differentiator/Differentiator.h"

__global__ void kernel(int *a) {
  *a *= *a;
}

// CHECK:    void kernel_grad(int *a, int *_d_a) {
//CHECK-NEXT:    int _t0 = *a;
//CHECK-NEXT:    *a *= *a;
//CHECK-NEXT:    {
//CHECK-NEXT:        *a = _t0;
//CHECK-NEXT:        int _r_d0 = *_d_a;
//CHECK-NEXT:        *_d_a = 0;
//CHECK-NEXT:        *_d_a += _r_d0 * *a;
//CHECK-NEXT:       *_d_a += *a * _r_d0;
//CHECK-NEXT:    }
//CHECK-NEXT: }

void fake_kernel(int *a) {
  *a *= *a;
}

__global__ void add_kernel(int *out, int *in) {
  int index = threadIdx.x;
  out[index] += in[index];
}

// CHECK:    void add_kernel_grad(int *out, int *in, int *_d_out, int *_d_in) {
//CHECK-NEXT:    int _d_index = 0;
//CHECK-NEXT:     int index0 = threadIdx.x;
//CHECK-NEXT:     int _t0 = out[index0];
//CHECK-NEXT:     out[index0] += in[index0];
//CHECK-NEXT:     {
//CHECK-NEXT:         out[index0] = _t0;
//CHECK-NEXT:         int _r_d0 = _d_out[index0];
//CHECK-NEXT:         _d_in[index0] += _r_d0;
//CHECK-NEXT:     }
//CHECK-NEXT: }

__global__ void add_kernel_2(int *out, int *in) {
  out[threadIdx.x] += in[threadIdx.x];
}

// CHECK:    void add_kernel_2_grad(int *out, int *in, int *_d_out, int *_d_in) {
//CHECK-NEXT:     int _t0 = out[threadIdx.x];
//CHECK-NEXT:     out[threadIdx.x] += in[threadIdx.x];
//CHECK-NEXT:     {
//CHECK-NEXT:         out[threadIdx.x] = _t0;
//CHECK-NEXT:         int _r_d0 = _d_out[threadIdx.x];
//CHECK-NEXT:         _d_in[threadIdx.x] += _r_d0;
//CHECK-NEXT:     }
//CHECK-NEXT: }

__global__ void add_kernel_3(int *out, int *in) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  out[index] += in[index];
}

// CHECK:    void add_kernel_3_grad(int *out, int *in, int *_d_out, int *_d_in) {
//CHECK-NEXT:    unsigned int _t1 = blockIdx.x;
//CHECK-NEXT:    unsigned int _t0 = blockDim.x;
//CHECK-NEXT:    int _d_index = 0;
//CHECK-NEXT:    int index0 = threadIdx.x + _t1 * _t0;
//CHECK-NEXT:    int _t2 = out[index0];
//CHECK-NEXT:    out[index0] += in[index0];
//CHECK-NEXT:    {
//CHECK-NEXT:        out[index0] = _t2;
//CHECK-NEXT:        int _r_d0 = _d_out[index0];
//CHECK-NEXT:        _d_in[index0] += _r_d0;
//CHECK-NEXT:    }
//CHECK-NEXT:}

__global__ void add_kernel_4(int *out, int *in) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < 5) {
    int sum = 0;
    // Each thread sums elements in steps of warpSize
    for (int i = index; i < 5; i += warpSize) {
        sum += in[i];
    }
    out[index] = sum;
  }
}

// CHECK: void add_kernel_4_grad(int *out, int *in, int *_d_out, int *_d_in) {
//CHECK-NEXT:    bool _cond0;
//CHECK-NEXT:    int _d_sum = 0;
//CHECK-NEXT:    int sum = 0;
//CHECK-NEXT:    unsigned long _t2;
//CHECK-NEXT:    int _d_i = 0;
//CHECK-NEXT:    int i = 0;
//CHECK-NEXT:    clad::tape<int> _t3 = {};
//CHECK-NEXT:    clad::tape<int> _t4 = {};
//CHECK-NEXT:    int _t5;
//CHECK-NEXT:    unsigned int _t1 = blockIdx.x;
//CHECK-NEXT:    unsigned int _t0 = blockDim.x;
//CHECK-NEXT:    int _d_index = 0;
//CHECK-NEXT:    int index0 = threadIdx.x + _t1 * _t0;
//CHECK-NEXT:    {
//CHECK-NEXT:        _cond0 = index0 < 5;
//CHECK-NEXT:        if (_cond0) {
//CHECK-NEXT:            sum = 0;
//CHECK-NEXT:            _t2 = 0UL;
//CHECK-NEXT:            for (i = index0; ; clad::push(_t3, i) , (i += warpSize)) {
//CHECK-NEXT:                {
//CHECK-NEXT:                    if (!(i < 5))
//CHECK-NEXT:                        break;
//CHECK-NEXT:                }
//CHECK-NEXT:                _t2++;
//CHECK-NEXT:                clad::push(_t4, sum);
//CHECK-NEXT:                sum += in[i];
//CHECK-NEXT:            }
//CHECK-NEXT:            _t5 = out[index0];
//CHECK-NEXT:            out[index0] = sum;
//CHECK-NEXT:        }
//CHECK-NEXT:    }
//CHECK-NEXT:    if (_cond0) {
//CHECK-NEXT:        {
//CHECK-NEXT:            out[index0] = _t5;
//CHECK-NEXT:            int _r_d2 = _d_out[index0];
//CHECK-NEXT:            _d_out[index0] = 0;
//CHECK-NEXT:            _d_sum += _r_d2;
//CHECK-NEXT:        }
//CHECK-NEXT:        {
//CHECK-NEXT:            for (;; _t2--) {
//CHECK-NEXT:                {
//CHECK-NEXT:                    if (!_t2)
//CHECK-NEXT:                        break;
//CHECK-NEXT:                }
//CHECK-NEXT:                {
//CHECK-NEXT:                    i = clad::pop(_t3);
//CHECK-NEXT:                    int _r_d0 = _d_i;
//CHECK-NEXT:                }
//CHECK-NEXT:                {
//CHECK-NEXT:                    sum = clad::pop(_t4);
//CHECK-NEXT:                    int _r_d1 = _d_sum;
//CHECK-NEXT:                    _d_in[i] += _r_d1;
//CHECK-NEXT:                }
//CHECK-NEXT:            }
//CHECK-NEXT:            _d_index += _d_i;
//CHECK-NEXT:        }
//CHECK-NEXT:    }
//CHECK-NEXT:}

__global__ void add_kernel_5(int *out, int *in) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < 5) {
        int sum = 0;
        // Calculate the total number of threads in the grid
        int totalThreads = blockDim.x * gridDim.x;
        // Each thread sums elements in steps of the total number of threads in the grid
        for (int i = index; i < 5; i += totalThreads) {
            sum += in[i];
        }
        out[index] = sum;
    }
}

// CHECK: void add_kernel_5_grad(int *out, int *in, int *_d_out, int *_d_in) {
//CHECK-NEXT:    bool _cond0;
//CHECK-NEXT:   int _d_sum = 0;
//CHECK-NEXT:    int sum = 0;
//CHECK-NEXT:    unsigned int _t2;
//CHECK-NEXT:    unsigned int _t3;
//CHECK-NEXT:    int _d_totalThreads = 0;
//CHECK-NEXT:    int totalThreads = 0;
//CHECK-NEXT:    unsigned long _t4;
//CHECK-NEXT:    int _d_i = 0;
//CHECK-NEXT:    int i = 0;
//CHECK-NEXT:    clad::tape<int> _t5 = {};
//CHECK-NEXT:    clad::tape<int> _t6 = {};
//CHECK-NEXT:    int _t7;
//CHECK-NEXT:    unsigned int _t1 = blockIdx.x;
//CHECK-NEXT:    unsigned int _t0 = blockDim.x;
//CHECK-NEXT:    int _d_index = 0;
//CHECK-NEXT:    int index0 = threadIdx.x + _t1 * _t0;
//CHECK-NEXT:    {
//CHECK-NEXT:        _cond0 = index0 < 5;
//CHECK-NEXT:        if (_cond0) {
//CHECK-NEXT:            sum = 0;
//CHECK-NEXT:            _t3 = blockDim.x;
//CHECK-NEXT:            _t2 = gridDim.x;
//CHECK-NEXT:            totalThreads = _t3 * _t2;
//CHECK-NEXT:            _t4 = 0UL;
//CHECK-NEXT:            for (i = index0; ; clad::push(_t5, i) , (i += totalThreads)) {
//CHECK-NEXT:                {
//CHECK-NEXT:                   if (!(i < 5))
//CHECK-NEXT:                       break;
//CHECK-NEXT:                }
//CHECK-NEXT:                _t4++;
//CHECK-NEXT:                clad::push(_t6, sum);
//CHECK-NEXT:                sum += in[i];
//CHECK-NEXT:            }
//CHECK-NEXT:            _t7 = out[index0];
//CHECK-NEXT:            out[index0] = sum;
//CHECK-NEXT:        }
//CHECK-NEXT:    }
//CHECK-NEXT:    if (_cond0) {
//CHECK-NEXT:        {
//CHECK-NEXT:            out[index0] = _t7;
//CHECK-NEXT:            int _r_d2 = _d_out[index0];
//CHECK-NEXT:            _d_out[index0] = 0;
//CHECK-NEXT:            _d_sum += _r_d2;
//CHECK-NEXT:        }
//CHECK-NEXT:        {
//CHECK-NEXT:            for (;; _t4--) {
//CHECK-NEXT:                {
//CHECK-NEXT:                    if (!_t4)
//CHECK-NEXT:                        break;
//CHECK-NEXT:                }
//CHECK-NEXT:                {
//CHECK-NEXT:                    i = clad::pop(_t5);
//CHECK-NEXT:                    int _r_d0 = _d_i;
//CHECK-NEXT:                    _d_totalThreads += _r_d0;
//CHECK-NEXT:                }
//CHECK-NEXT:                {
//CHECK-NEXT:                    sum = clad::pop(_t6);
//CHECK-NEXT:                    int _r_d1 = _d_sum;
//CHECK-NEXT:                    _d_in[i] += _r_d1;
//CHECK-NEXT:                }
//CHECK-NEXT:            }
//CHECK-NEXT:            _d_index += _d_i;
//CHECK-NEXT:        }
//CHECK-NEXT:    }
//CHECK-NEXT:}

#define TEST(F, grid, block, shared_mem, use_stream, x, dx, N)              \
  {                                                                         \
    int *fives = (int*)malloc(N * sizeof(int));                             \
    for(int i = 0; i < N; i++) {                                            \
      fives[i] = 5;                                                         \
    }                                                                       \
    int *ones = (int*)malloc(N * sizeof(int));                              \
    for(int i = 0; i < N; i++) {                                            \
      ones[i] = 1;                                                          \
    }                                                                       \
    hipMemcpy(x, fives, N * sizeof(int), hipMemcpyHostToDevice);          \
    hipMemcpy(dx, ones, N * sizeof(int), hipMemcpyHostToDevice);          \
    auto test = clad::gradient(F);                                          \
    if constexpr (use_stream) {                                             \
      hipStream_t cudaStream;                                              \
      hipStreamCreate(&cudaStream);                                        \
      test.execute_kernel(grid, block, shared_mem, cudaStream, x, dx);      \
    }                                                                       \
    else {                                                                  \
      test.execute_kernel(grid, block, x, dx);                              \
    }                                                                       \
    hipDeviceSynchronize();                                                \
    int *res = (int*)malloc(N * sizeof(int));                               \
    hipMemcpy(res, dx, N * sizeof(int), hipMemcpyDeviceToHost);           \
    for (int i = 0; i < (N - 1); i++) {                                     \
      printf("%d, ", res[i]);                                               \
    }                                                                       \
    printf("%d\n", res[N-1]);                                               \
    free(fives);                                                            \
    free(ones);                                                             \
    free(res);                                                              \
  }


#define TEST_2(F, grid, block, shared_mem, use_stream, args, y, x, dy, dx, N) \
  {                                                                           \
    int *fives = (int*)malloc(N * sizeof(int));                               \
    for(int i = 0; i < N; i++) {                                              \
      fives[i] = 5;                                                           \
    }                                                                         \
    int *zeros = (int*)malloc(N * sizeof(int));                               \
    for(int i = 0; i < N; i++) {                                              \
      zeros[i] = 0;                                                           \
    }                                                                         \
    hipMemcpy(x, fives, N * sizeof(int), hipMemcpyHostToDevice);            \
    hipMemcpy(y, zeros, N * sizeof(int), hipMemcpyHostToDevice);            \
    hipMemcpy(dy, fives, N * sizeof(int), hipMemcpyHostToDevice);           \
    hipMemcpy(dx, zeros, N * sizeof(int), hipMemcpyHostToDevice);           \
    auto test = clad::gradient(F, args);                                      \
    if constexpr (use_stream) {                                               \
      hipStream_t cudaStream;                                                \
      hipStreamCreate(&cudaStream);                                          \
      test.execute_kernel(grid, block, shared_mem, cudaStream, y, x, dy, dx); \
    }                                                                         \
    else {                                                                    \
      test.execute_kernel(grid, block, y, x, dy, dx);                         \
    }                                                                         \
    hipDeviceSynchronize();                                                  \
    int *res = (int*)malloc(N * sizeof(int));                                 \
    hipMemcpy(res, dx, N * sizeof(int), hipMemcpyDeviceToHost);             \
    for (int i = 0; i < (N - 1); i++) {                                       \
      printf("%d, ", res[i]);                                                 \
    }                                                                         \
    printf("%d\n", res[N-1]);                                                 \
    free(fives);                                                              \
    free(zeros);                                                              \
    free(res);                                                                \
  }


int main(void) {
  int *a, *d_a;
  hipMalloc(&a, sizeof(int));
  hipMalloc(&d_a, sizeof(int));

  TEST(kernel, dim3(1), dim3(1), 0, false, a, d_a, 1); // CHECK-EXEC: 10
  TEST(kernel, dim3(1), dim3(1), 0, true, a, d_a, 1); // CHECK-EXEC: 10

  auto error = clad::gradient(fake_kernel); 
  error.execute_kernel(dim3(1), dim3(1), a, d_a); // CHECK-EXEC: Use execute() for non-global CUDA kernels

  auto test = clad::gradient(kernel);
  test.execute(a, d_a); // CHECK-EXEC: Use execute_kernel() for global CUDA kernels

  hipFree(a);
  hipFree(d_a);


  int *dummy_in, *dummy_out, *d_out, *d_in;
  hipMalloc(&dummy_in, 5 * sizeof(int));
  hipMalloc(&dummy_out, 5 * sizeof(int));
  hipMalloc(&d_out, 5 * sizeof(int));
  hipMalloc(&d_in, 5 * sizeof(int));

  TEST_2(add_kernel, dim3(1), dim3(5, 1, 1), 0, false, "in, out", dummy_out, dummy_in, d_out, d_in, 5); // CHECK-EXEC: 5, 5, 5, 5, 5
  TEST_2(add_kernel_2, dim3(1), dim3(5, 1, 1), 0, true, "in, out", dummy_out, dummy_in, d_out, d_in, 5); // CHECK-EXEC: 5, 5, 5, 5, 5
  TEST_2(add_kernel_3, dim3(5, 1, 1), dim3(1), 0, false, "in, out", dummy_out, dummy_in, d_out, d_in, 5); // CHECK-EXEC: 5, 5, 5, 5, 5
  TEST_2(add_kernel_4, dim3(1), dim3(5, 1, 1), 0, false, "in, out", dummy_out, dummy_in, d_out, d_in, 5); // CHECK-EXEC: 5, 5, 5, 5, 5
  TEST_2(add_kernel_5, dim3(2, 1, 1), dim3(1), 0, false, "in, out", dummy_out, dummy_in, d_out, d_in, 5); // CHECK-EXEC: 5, 5, 5, 5, 5

  hipFree(dummy_in);
  hipFree(dummy_out);
  hipFree(d_out);
  hipFree(d_in);

  return 0;
}
